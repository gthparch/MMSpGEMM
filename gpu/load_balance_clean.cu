#include "hip/hip_runtime.h"
#include <moderngpu/kernel_compact.hxx>
#include <moderngpu/kernel_load_balance.hxx>
#include <moderngpu/kernel_scan.hxx>
#include <moderngpu/kernel_reduce.hxx>
#include <moderngpu/cta_load_balance.hxx>
#include <moderngpu/cta_scan.hxx>
#include <moderngpu/cta_mergesort.hxx>

#include <hipcub/hipcub.hpp>

#include <iostream>
#include <fstream>
#include <random>
#include <cstring>
#include <chrono>
#include <map>
#include <unordered_map>

#include "MatrixMarket.h"
#include "DeviceMatrix.h"

// must match BLOCK_SIZE used to generate blocks in ../load-balance-test.cpp
#define BLOCK_SIZE          2048
#define ITEMS_PER_THREAD    16
#define NUM_THREADS         128

// TODO: Put this in a proper header file to share
class TSplit
{
public:
    int a_row;
    int bp;
    int b_col;
};


struct CustomLess
{
    template <typename DataType>
    __device__ bool operator()(const DataType& lhs, const DataType& rhs) { return lhs < rhs; }
};


// XXX: TODO: Fix using 32-bit ints for pointers in kernel
struct block_output_row
{
    uint64_t start;
    /*
    uint64_t key_in;
    uint64_t key_out;
    uint32_t size;
    */
    float carry_out;
};


void CheckCuda(hipError_t e)
{
    if (e != hipSuccess) {
        std::cerr << "CUDA ERROR! " << hipGetErrorName(e) << " : " << hipGetErrorString(e) << std::endl;
        exit(1);
    }
}


__device__ int lower_bound_search(int *arr, int q, int N)
{
    int mid;

    int low = 0;
    int high = N;

    while (low < high)
    {
        mid = low + (high - low) / 2;

        if (q <= arr[mid])
            high = mid;
        else
            low = mid + 1;
    }

    if (low < N && arr[low] < q)
        low++;

    return low;
}

__device__ int upper_bound_search(int *arr, int q, int N)
{
    int mid;

    int low = 0;
    int high = N;

    while (low < high)
    {
        mid = low + (high - low) / 2;

        if (q >= arr[mid])
            low = mid + 1;
        else
            high = mid;
    }

    if (low < N && arr[low] <= q)
        low++;

    return low;
}


__global__ void cuda_build_thread_splits_opt(const int *lb_data, const int *lb_block_ptrs, const int *ARowPtrs, const int *AColIdx,
                                             const float *AmCSRVals, const int *BmRowPtrs, const int *BmColIdx, const float *BmCSRVals,
                                             int nblocks, int column_bits, float *out_test)
{
    int block = blockIdx.x;

    int cur_bp = lb_block_ptrs[block];
    int next_bp = lb_block_ptrs[block+1];
    int start_row = lb_data[cur_bp];
    int end_row = lb_data[next_bp];

    // these two can be a union to save space, but check that the cumsum dependencies work
    __shared__ int row_counts[NUM_THREADS];
    __shared__ int cum_row_counts[NUM_THREADS];     // should be one more than NUM_THREADS since this is an exclusive scan
    __shared__ int shared_cum_sizes[BLOCK_SIZE];

    uint32_t thread_keys[ITEMS_PER_THREAD];
    float thread_vals[ITEMS_PER_THREAD];

    int segment_sizes[ITEMS_PER_THREAD];            // this can be higher; this limits the number of segments we allow

    int nrows = end_row - start_row + 1;

    // XXX: Can't handle blocks with more than NUM_THREADS rows ...
    if (threadIdx.x < nrows)
    {
        int row = start_row + threadIdx.x;
        row_counts[threadIdx.x] = ARowPtrs[row+1] - ARowPtrs[row];
    }
    __syncthreads();

    // This sucks but the overhead of load balancing this seems like it will outweight the benefit
    if (threadIdx.x == 0)
    {
        shared_cum_sizes[0] = 0;
        cum_row_counts[0] = 0;
        for (int i=0; i < nrows; i++)
            cum_row_counts[i+1] = row_counts[i] + cum_row_counts[i];
    }
    __syncthreads();

    // XXX: assert total count is less than some reasonable number
    // segments_per_thread < ITEMS_PER_THREAD

    int segments_per_thread = (cum_row_counts[nrows] / NUM_THREADS) + 1;
    int s = threadIdx.x * segments_per_thread;
    int last_ss = 0;
    int seg = 0;
    if (s < cum_row_counts[nrows])
    {
        // XXX: Slow linear scan; not expecting many rows (on the order of 5-10)
        int p = 1;
        // XXX: Not checking boundary condition since we checked s in the condition above
        while (cum_row_counts[p] <= s) p++;
        int row = p - 1;
        int rank = s - cum_row_counts[row];       // rank is now the column in that row that we're computing seg size for

        int col_start = ARowPtrs[row + start_row];
        for (seg = 0; seg < segments_per_thread; seg++)
        {
            // every row has to have one non-zero (else this should be a while)
            if (seg + s == cum_row_counts[row+1]) {
                row++;
                // check row is in range
                if (row == nrows)
                    break;

                // if start_row or end_row, take the sizes from splits
                col_start = ARowPtrs[row + start_row];
                rank = 0;
            }

            int brow = AColIdx[col_start + rank] - 1;
            int bstart = BmRowPtrs[brow];
            int bend = BmRowPtrs[brow+1];

            if (row == 0)
                bstart += lb_data[cur_bp + 3 + rank];
            if (row == nrows-1)
                bend = BmRowPtrs[brow] + lb_data[next_bp + 3 + rank];

            segment_sizes[seg] = bend - bstart + last_ss;
            last_ss = segment_sizes[seg];
            rank++;
        }
    }
    __syncthreads();

    // May want to set up temp storage explicitly to reuse it
    // Can we do an exclusive sum with only the first s < cumrow threads?
    hipcub::BlockScan<int, NUM_THREADS>().ExclusiveSum(last_ss, last_ss);

    // Now write out shared_cum_sizes
    if (s < cum_row_counts[nrows])
    {
        for (int i = 0; i < seg; i++) {
            shared_cum_sizes[s + i + 1] = segment_sizes[i] + last_ss;
        }
    }
    __syncthreads();

    /*
    if (shared_cum_sizes[cum_row_counts[nrows]] != BLOCK_SIZE)
    {
        printf("ERROR in block %d not equal BLOCK_SIZE\n", block);
    }
    */

    // each thread finds itself in the scan above and copies ITEMS_PER_THREAD starting from that location
    int copy_start = upper_bound_search(shared_cum_sizes, threadIdx.x * ITEMS_PER_THREAD, cum_row_counts[nrows]) - 1;
    int row_start = upper_bound_search(cum_row_counts, copy_start, nrows) - 1;
    int row_rank = copy_start - cum_row_counts[row_start];
    int col_rank = threadIdx.x * ITEMS_PER_THREAD - shared_cum_sizes[copy_start];

    int ap = ARowPtrs[row_start + start_row] + row_rank;
    int row_end = ARowPtrs[row_start + start_row + 1];
    float acoeff = AmCSRVals[ap];
    int acol = AColIdx[ap] - 1;         // also == brow
    int initial_ap = ARowPtrs[row_start + start_row];      // not adding row_rank
    int bp = BmRowPtrs[acol] + col_rank;
    if (row_start == 0)
        bp += lb_data[cur_bp + 3 + (ap - initial_ap)];

    for (int i=0; i < ITEMS_PER_THREAD; i++)
    {
        while (threadIdx.x * ITEMS_PER_THREAD + i == shared_cum_sizes[copy_start + 1])
        {
            // bump the next ap
            ap++;
            if (ap == row_end) {
                row_start++;
                row_end = ARowPtrs[row_start + start_row + 1];
            }
            acoeff = AmCSRVals[ap];
            bp = BmRowPtrs[AColIdx[ap] - 1];
            copy_start++;
            if (row_start == 0) {
                bp += lb_data[cur_bp + 3 + (ap - initial_ap)];
            }
        }
        thread_keys[i] = (row_start << column_bits) | BmColIdx[bp];
//        thread_vals[i] = BmCSRVals[bp] * acoeff;
        out_test[(blockIdx.x * NUM_THREADS + threadIdx.x) * ITEMS_PER_THREAD + i] = BmCSRVals[bp];
        bp++;
    }
}


__global__ void cuda_build_thread_splits(const int *lb_data, const int *lb_block_ptrs, const int *BmRowPtrs, const int *BmColIdx, TSplit *thread_splits, int nblocks)
{
    int block = blockIdx.x * blockDim.x + threadIdx.x;
    if (block > nblocks)
        return;

    int cur_bp = lb_block_ptrs[block];
    int next_bp = lb_block_ptrs[block+1];
    int start_row = lb_data[cur_bp];
    int end_row = lb_data[next_bp];
    int copy_count(0);
    int ti(0);

    for (int row = start_row; row <= end_row; row++)
    {
        int coeff_start = BmRowPtrs[row];
        int coeff_end = BmRowPtrs[row+1];
        for (int bp = coeff_start; bp < coeff_end; bp++)
        {
            int brow = BmColIdx[bp]-1;
            int seg_start = BmRowPtrs[brow];
            int seg_end = BmRowPtrs[brow+1];
            if (row == start_row)
                seg_start += lb_data[cur_bp + 3 + (bp - coeff_start)];
            if (row == end_row)
                seg_end = BmRowPtrs[brow] + lb_data[next_bp + 3 + (bp - coeff_start)];

            int count = seg_end - seg_start;
            /* Original */
            for (int i = 0; i < count; i++)
            {
                if (copy_count % ITEMS_PER_THREAD == 0) {
                    thread_splits[block * NUM_THREADS + ti] = {row, bp, seg_start + i};
                    ti++;
                }
                copy_count++;
            }

            // This is really no faster than the original code :-(
            // Also unverified
            /*
            if (copy_count % ITEMS_PER_THREAD == 0)
            {
                thread_splits[block * NUM_THREADS + ti] = {row, bp, seg_start};
                ti++;
            }
            int i = ITEMS_PER_THREAD - (copy_count % ITEMS_PER_THREAD);
            while (i < count) {
                thread_splits[block * NUM_THREADS + ti] = {row, bp, seg_start + i};
                ti++;
                i += ITEMS_PER_THREAD;
            }
            copy_count += count;
            */
        }
    }
}


//template <int COLUMN_BITS=COLUMN_BITS, int TOTAL_BITS=TOTAL_BITS>
__global__ void cuda_load_block_coop(const int *AmRowPtrs, const int *AmColIdx, const float *AmCSRVals,
                                     const int *BmRowPtrs, const int *BmColIdx, const float *BmCSRVals,
                                     const int *lb_data, const int *lb_block_ptrs, const TSplit* lb_thread_splits,
                                     uint32_t *out_keys, float *out_vals, int *atomic_p, block_output_row *out_meta, int *out_sizes,
                                     int column_bits, float *out_test)
{
    int block = blockIdx.x;
    int cur_block_ptr = lb_block_ptrs[block];
    int next_block_ptr = lb_block_ptrs[block+1];
    int end_row = lb_data[next_block_ptr];
    int start_row = lb_data[cur_block_ptr];
    int total_bits = column_bits + (32 - __clz(end_row-start_row));

    typedef hipcub::BlockRadixSort<uint32_t, NUM_THREADS, ITEMS_PER_THREAD, float, 4, true> BlockRadixSort;
    __shared__ union {
        typename BlockRadixSort::TempStorage block_radix_storage;
    } smem;
    uint32_t thread_keys[ITEMS_PER_THREAD];
    float thread_vals[ITEMS_PER_THREAD];

    TSplit split = lb_thread_splits[block * NUM_THREADS + threadIdx.x];
    int row_end = AmRowPtrs[split.a_row + 1];
    int brow = AmColIdx[split.bp] - 1;
    int seg_end = BmRowPtrs[brow+1];

    int coeff_start = AmRowPtrs[split.a_row];
    if (split.a_row == end_row)
        seg_end = BmRowPtrs[brow] + lb_data[next_block_ptr + 3 + (split.bp - coeff_start)];
    float Acoeff = AmCSRVals[split.bp];

    for (int i = 0; i < ITEMS_PER_THREAD; i++)
    {
        while (split.b_col >= seg_end)
        {
            split.bp++;
            if (split.bp >= row_end) {
                split.a_row++;
                split.bp = AmRowPtrs[split.a_row];
                row_end = AmRowPtrs[split.a_row + 1];
            }
            Acoeff = AmCSRVals[split.bp];
            brow = AmColIdx[split.bp] - 1;
            split.b_col = BmRowPtrs[brow];
            seg_end = BmRowPtrs[brow+1];
            if (split.a_row == start_row) {
                int coeff_start = AmRowPtrs[split.a_row];
                split.b_col += lb_data[cur_block_ptr + 3 + (split.bp - coeff_start)];
            }
            if (split.a_row == end_row) {
                int coeff_start = AmRowPtrs[split.a_row];
                seg_end = BmRowPtrs[brow] + lb_data[next_block_ptr + 3 + (split.bp - coeff_start)];
            }
        }

        thread_keys[i] = ((split.a_row - start_row) << column_bits) | BmColIdx[split.b_col];
//        thread_vals[i] = BmCSRVals[split.b_col] * Acoeff;
        out_test[(blockIdx.x * NUM_THREADS + threadIdx.x) * ITEMS_PER_THREAD + i] = BmCSRVals[split.b_col];

        split.b_col++;
    }

    return;

    __syncthreads();

    BlockRadixSort(smem.block_radix_storage).Sort(thread_keys, thread_vals, 0, total_bits);

    __syncthreads();


    // debugging post-sort
    /*
    bool debug = false;
    for (int i = 0; i < ITEMS_PER_THREAD; i++) {
        if ((thread_keys[i] >> column_bits) + start_row == 358 && (thread_keys[i] & ((1 << column_bits) - 1)) == 534) {
            debug = true;
            printf("block %d (post-sort), tid: %d, i = %d, val = %f\n", block, threadIdx.x, i, thread_vals[i]);
        }
//        output_vals[threadIdx.x * ITEMS_PER_THREAD + i] = thread_vals[i];
//        output_keys[threadIdx.x * ITEMS_PER_THREAD + i] = thread_keys[i];
    }
    */



/************************************************
 * Reduction phase
 */
    #define FULL_MASK   0xffffffff
    int warp = threadIdx.x / 32;
    int lane = threadIdx.x & 31;

    // XXX: Hard-coded 4 warps
    // Could combine these in the union below ...?
	__shared__ int delta_shared[4 + NUM_THREADS];	
    __shared__ float carry_out[NUM_THREADS];
    __shared__ uint32_t block_keys[4 + 1];       // reuse delta_shared?
    block_keys[4] = 0xffffffff;

    if (lane == 0)
        block_keys[warp] = thread_keys[0];

    int p = 0;
    for (int i=1; i < ITEMS_PER_THREAD; i++)
    {
        if (thread_keys[p] == thread_keys[i]) {
            thread_vals[p] += thread_vals[i];
        }
        else {
            p++;
            thread_keys[p] = thread_keys[i];
            thread_vals[p] = thread_vals[i];
        }
    }
    carry_out[threadIdx.x] = thread_vals[p];
    __syncthreads();        // XXX: Leave me

    uint32_t neighbor_key = __shfl_down_sync(FULL_MASK, thread_keys[0], 1);
    if (lane == 31) {
        neighbor_key = block_keys[warp+1];
    }
//    __syncthreads();      // Commenting seems OK
    // what happens to the very last warp carry_out?
    bool rseg_end = (p > 0);
    if (thread_keys[ITEMS_PER_THREAD-1] != neighbor_key) {
        p++;
        if (threadIdx.x < (NUM_THREADS-1)) {
            rseg_end = true;
            carry_out[threadIdx.x] = 0.0;
        }
    }
    __syncthreads();

    uint warp_mask = FULL_MASK >> (31 - lane);
    uint cta_mask = 0x7fffffff >> (31 - lane);

    uint warp_bits = __ballot_sync(FULL_MASK, rseg_end);
    delta_shared[warp] = warp_bits;
    __syncthreads();

    const int NumWarps = NUM_THREADS / 32;
    if (threadIdx.x < NumWarps) {
        uint cta_bits = __ballot_sync(0xf, 0 != delta_shared[threadIdx.x]);
        int warpSegment = 31 - __clz(cta_mask & cta_bits);
        int start = (-1 != warpSegment) ? 
            (31 - __clz(delta_shared[warpSegment]) + 32 * warpSegment) : 0;
        delta_shared[NumWarps + threadIdx.x] = start;
    }
    __syncthreads();
 
    // Find the closest flag to the left of this thread within the warp.
    // Include the flag for this thread.
    int start = 31 - __clz(warp_mask & warp_bits);
    if(-1 != start) start += ~31 & threadIdx.x;
    else start = delta_shared[NumWarps + warp];
    __syncthreads();
 
    uint tid_delta = threadIdx.x - start;

    for (int offset = 1; offset < NUM_THREADS; offset += offset)
    {
        if (tid_delta >= offset)
            carry_out[threadIdx.x] += carry_out[threadIdx.x - offset];
        __syncthreads();
    }

    // apply carry_out ...
//    if (p > 0 && threadIdx.x > 0)
    if (threadIdx.x > 0) {
        thread_vals[0] += carry_out[threadIdx.x - 1];
    }

    // do need inclusive sum 
    typedef hipcub::BlockScan<int, NUM_THREADS> BlockScanWriteOut;
    __shared__ typename BlockScanWriteOut::TempStorage bswo_storage;

    int start_p, total_p;
    BlockScanWriteOut(bswo_storage).ExclusiveSum(p, start_p, total_p);

    // TODO: coop write worth it?
    __shared__ int shared_p;
    if (threadIdx.x == 0) {
        shared_p = atomicAdd(atomic_p, total_p);
    }
    __syncthreads();

    if (threadIdx.x == NUM_THREADS-1) {
        out_meta[block].start = shared_p;
        if (lb_data[cur_block_ptr + 2])
            total_p--;      // may want to do this before block above so that we only write total_p - 1
        out_sizes[block] = total_p;
        if (!rseg_end)
            out_meta[block].carry_out = thread_vals[0];
        else
            out_meta[block].carry_out = carry_out[threadIdx.x];
    }

    start_p += shared_p;
    for (int i=0; i < p; i++) {
        out_keys[start_p + i] = thread_keys[i];
        out_vals[start_p + i] = thread_vals[i];
    }


/*
 * End reduction phase
 **************************************************************/
}


int main(int argc, char **argv)
{
    mgpu::standard_context_t context;

    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " <mtx file>" << std::endl;
        exit(1);
    }

    MatrixMarket matA { argv[1] };
    MatrixMarket& matB = matA;
    DeviceMatrix dmA(matA, context);
    DeviceMatrix& dmB = dmA;

    int column_bits = int(logf((float)matA.mRows) / logf(2.0)) + 1;
    std::cout << "column_bits = " << column_bits << std::endl;

    std::ifstream if_data("../lb_data.bin", std::ifstream::binary);
    std::ifstream if_block_ptr("../lb_block_ptrs.bin", std::ifstream::binary);
//    std::ifstream if_thread_splits("../lb_thread_splits.bin", std::ifstream::binary);

    if_data.seekg(0, if_data.end);
    if_block_ptr.seekg(0, if_block_ptr.end);
//    if_thread_splits.seekg(0, if_thread_splits.end);
    int lb_data_size = if_data.tellg();
    int lb_block_ptrs_size = if_block_ptr.tellg();
//    int lb_thread_splits_size = if_thread_splits.tellg();
    if_data.seekg(0, if_data.beg);
    if_block_ptr.seekg(0, if_block_ptr.beg);
//    if_thread_splits.seekg(0, if_thread_splits.beg);

    std::cout << "lb_data size = " << lb_data_size << std::endl;
    std::cout << "lb_block_ptr size = " << lb_block_ptrs_size << std::endl;
//    std::cout << "lb_thread_splits size = " << lb_thread_splits_size << std::endl;

    std::vector<int> lb_data, lb_block_ptrs;
//    std::vector<std::vector<TSplit>> lb_thread_splits;
    lb_data.resize(lb_data_size >> 2);
    lb_block_ptrs.resize(lb_block_ptrs_size >> 2);
    /*
    lb_thread_splits.resize(lb_block_ptrs_size >> 2);
    for (int i=0; i < lb_thread_splits.size()-1; i++) {
        lb_thread_splits[i].resize(128);
        if_thread_splits.read((char *)lb_thread_splits[i].data(), sizeof(TSplit) * 128);
    }
    */

    if_data.read((char *)lb_data.data(), lb_data_size);
    if_block_ptr.read((char *)lb_block_ptrs.data(), lb_block_ptrs_size);

    std::cout << "read lb_data = " << lb_data.size() << std::endl;
    std::cout << "read lb_block_ptrs = " << lb_block_ptrs.size() << std::endl;
//    std::cout << "read lb_thread_splits" << std::endl;

    /*
    for (int i=1; i < lb_block_ptrs.size(); i++) {
        if (lb_data[lb_block_ptrs[i]] == lb_data[lb_block_ptrs[i-1]]) {
            std::cout << "single row: " << i << std::endl;
            break;
        }
    }
    */

    /*
    std::vector<TSplit> lb_flat_splits;
    for (int i = 0; i < lb_thread_splits.size()-1; i++)
    {
        for (int j = 0; j < NUM_THREADS; j++)
            lb_flat_splits.push_back(lb_thread_splits[i][j]);
    }
    */
    mgpu::mem_t<int> d_lb_data = mgpu::to_mem(lb_data, context);
    mgpu::mem_t<int> d_lb_block_ptrs = mgpu::to_mem(lb_block_ptrs, context);
//    mgpu::mem_t<TSplit> d_flat_tsplits = mgpu::to_mem(lb_flat_splits, context);

    std::vector<float> out_buffer;
    std::vector<int> out_buffer_keys;

    std::vector<uint32_t> out_keys, final_keys_rows, final_keys_cols;
    std::vector<int> segments;
    std::vector<float> out_vals, final_vals;
    std::vector<block_output_row> out_meta;
    std::vector<int> out_sizes;
    out_meta.resize(lb_block_ptrs.size());
    out_sizes.resize(lb_block_ptrs.size());
    out_keys.resize(200000000);
    out_vals.resize(200000000);
    final_keys_rows.resize(200000000);
    final_keys_cols.resize(200000000);
    final_vals.resize(200000000);
    segments.resize(lb_block_ptrs.size());
    std::vector<int> atomic_p;
    atomic_p.resize(2);

    out_buffer.resize(BLOCK_SIZE);
    out_buffer_keys.resize(BLOCK_SIZE);
    mgpu::mem_t<float> d_output = mgpu::to_mem(out_buffer, context);
    mgpu::mem_t<int> d_output_keys = mgpu::to_mem(out_buffer_keys, context);

    mgpu::mem_t<uint32_t> d_out_keys = mgpu::to_mem(out_keys, context);
    mgpu::mem_t<float> d_out_vals = mgpu::to_mem(out_vals, context);
    mgpu::mem_t<int> d_atomic_p = mgpu::to_mem(atomic_p, context);
    mgpu::mem_t<block_output_row> d_out_meta = mgpu::to_mem(out_meta, context);
    mgpu::mem_t<int> d_out_sizes = mgpu::to_mem(out_sizes, context);

    mgpu::mem_t<uint32_t> d_final_keys_rows = mgpu::to_mem(final_keys_rows, context);
    mgpu::mem_t<uint32_t> d_final_keys_cols = mgpu::to_mem(final_keys_cols, context);
    mgpu::mem_t<float> d_final_vals = mgpu::to_mem(final_vals, context);
    mgpu::mem_t<int> d_segments = mgpu::to_mem(segments, context);

    mgpu::mem_t<int> total_count(1, context);

    // row counts; need prefix sum to finalize
    uint32_t *d_final_row_counts_raw;
    hipMalloc(&d_final_row_counts_raw, sizeof(uint32_t) * matA.mRows);

    int *d_unique_out, *d_counts_out, *d_num_runs_out;
    hipMalloc(&d_unique_out, sizeof(int) * matA.mRows);
    hipMalloc(&d_counts_out, sizeof(int) * matA.mRows);
    hipMemset(d_counts_out, 0x0, sizeof(int) * matA.mRows);
    hipMalloc(&d_num_runs_out, sizeof(int) * 1);

    int *d_counts_out2;
    int *h_row_ptrs = new int[matA.mRows];
    hipMalloc(&d_counts_out2, matA.mRows * sizeof(int));

    hipEvent_t start_evt, stop_tsplit_evt, stop_compute_evt;
    hipEvent_t stop_shuffle_evt, stop_carries_evt, stop_rle_evt;
    float elapsed;

    CheckCuda(hipEventCreate(&start_evt));
    CheckCuda(hipEventCreate(&stop_tsplit_evt));
    CheckCuda(hipEventCreate(&stop_compute_evt));
    CheckCuda(hipEventCreate(&stop_shuffle_evt));
    CheckCuda(hipEventCreate(&stop_carries_evt));
    CheckCuda(hipEventCreate(&stop_rle_evt));

    mgpu::mem_t<float> out_test(lb_block_ptrs.size() * NUM_THREADS * ITEMS_PER_THREAD, context);
#if 1
    TSplit *d_flat_tsplits;
    hipMalloc(&d_flat_tsplits, sizeof(TSplit) * lb_block_ptrs.size() * NUM_THREADS);
    hipDeviceSynchronize();
    int nblocks = ((lb_block_ptrs.size()-1) / NUM_THREADS) + 1;
    std::cout << "nblocks = " << nblocks << ", nthreads = " << NUM_THREADS << ", " << lb_block_ptrs.size()-1 << std::endl;
    hipEventRecord(start_evt, 0);
//    cuda_build_thread_splits<<<nblocks, NUM_THREADS>>>(d_lb_data.data(), d_lb_block_ptrs.data(), dmB.raw.d_row_ptrs, dmB.raw.d_col_idx, d_flat_tsplits, lb_block_ptrs.size()-1);
    cuda_build_thread_splits_opt<<<lb_block_ptrs.size()-1, NUM_THREADS>>>(d_lb_data.data(), d_lb_block_ptrs.data(), dmA.raw.d_row_ptrs, dmA.raw.d_col_idx, dmA.raw.d_values, dmB.raw.d_row_ptrs, dmB.raw.d_col_idx, dmB.raw.d_values, lb_block_ptrs.size()-1, column_bits, out_test.data());
    hipEventRecord(stop_tsplit_evt, 0);
    hipEventSynchronize(stop_tsplit_evt);
    hipEventElapsedTime(&elapsed, start_evt, stop_tsplit_evt);
    std::cout << "Thread splits elapsed: " << elapsed << std::endl;

    std::vector<float> read_out_test = mgpu::from_mem(out_test);
    std::ofstream ofsx("out_test.bin");
    ofsx.write((const char *)read_out_test.data(), sizeof(float) * read_out_test.size());
    exit(0);
#else
    hipEventRecord(start_evt, 0);
#endif


    /*
    int numBlocks;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks, cuda_load_block_coop, NUM_THREADS, 2048 * 4);
    std::cout << "max occupancy(?) = " << numBlocks << std::endl;
    */

//    auto start = std::chrono::system_clock::now();
    // Block multiplies
    cuda_load_block_coop<<<lb_block_ptrs.size()-1, NUM_THREADS>>>(dmA.raw.d_row_ptrs, dmA.raw.d_col_idx, dmA.raw.d_values,
                                             dmB.raw.d_row_ptrs, dmB.raw.d_col_idx, dmB.raw.d_values,
                                             d_lb_data.data(), d_lb_block_ptrs.data(), d_flat_tsplits, d_out_keys.data(),
                                             d_out_vals.data(), d_atomic_p.data(),
                                             d_out_meta.data(), d_out_sizes.data(), column_bits, out_test.data());

    hipEventRecord(stop_compute_evt, 0);
    hipEventSynchronize(stop_compute_evt);
//    CheckCuda(hipDeviceSynchronize());
//    auto finish = std::chrono::system_clock::now();
//    double elapsed = std::chrono::duration<double, std::milli>(finish - start).count();
    hipEventElapsedTime(&elapsed, start_evt, stop_compute_evt);
    std::cout << "Compute elapsed: " << elapsed << std::endl;

    /*
    std::vector<float> read_out_test = mgpu::from_mem(out_test);
    std::ofstream ofsx("out_test.ref.bin");
    ofsx.write((const char *)read_out_test.data(), sizeof(float) * read_out_test.size());
    exit(0);
    */


    // (exclusive) prefix sum scan segments; needed for the shuffle gather and applying the carries
    mgpu::scan(d_out_sizes.data(), lb_block_ptrs.size()-1, d_segments.data(), mgpu::plus_t<int>(), total_count.data(), context);
    int total_count_h = mgpu::from_mem(total_count)[0];

    block_output_row *p = d_out_meta.data();
    float *d_out_vals_raw = d_out_vals.data();
    uint32_t *d_out_keys_raw = d_out_keys.data();
    float *d_final_vals_raw = d_final_vals.data();
    uint32_t *d_final_keys_rows_raw = d_final_keys_rows.data();
    uint32_t *d_final_keys_cols_raw = d_final_keys_cols.data();
    const int *d_lb_data_data = d_lb_data.data();
    const int *d_lb_block_ptrs_data = d_lb_block_ptrs.data();
    // Shuffle gather and split out rows and columns into two arrays
    // (TODO: See if computing the row indices in the kernel is faster and/or doing a symbolic pass for this first)
    mgpu::transform_lbs([=]MGPU_DEVICE(int index, int seg, int rank) {
        int pp = p[seg].start + rank;
        int start_row = d_lb_data_data[d_lb_block_ptrs_data[seg]];
        uint32_t key = d_out_keys_raw[pp];
        d_final_keys_cols_raw[index] = key & ((1 << column_bits) - 1);
        d_final_keys_rows_raw[index] = (key >> column_bits) + start_row;
        d_final_vals_raw[index] = d_out_vals_raw[pp];
    }, total_count_h, d_segments.data(), lb_block_ptrs.size()-1, context);

    hipEventRecord(stop_shuffle_evt, 0);

    // Apply carries
    int *d_segments_data = d_segments.data();
    mgpu::transform([=]MGPU_DEVICE(int i) {
        int curblock = d_lb_block_ptrs_data[i];
        int overflow = d_lb_data_data[curblock + 2];
        if (overflow)
            atomicAdd(d_final_vals_raw + d_segments_data[i+1], p[i].carry_out);
    }, lb_block_ptrs.size()-2, context);

    hipEventRecord(stop_carries_evt, 0);

    void *temp_storage = NULL;
    size_t temp_storage_bytes;
    hipcub::DeviceRunLengthEncode::Encode(temp_storage, temp_storage_bytes, d_final_keys_rows_raw, (int*)0, (int*)0, (int*)0, total_count_h);
    std::cout << "temp_storage_bytes = " << temp_storage_bytes << std::endl;
    CheckCuda(hipMalloc(&temp_storage, temp_storage_bytes));

    CheckCuda(hipcub::DeviceRunLengthEncode::Encode(temp_storage, temp_storage_bytes, d_final_keys_rows_raw, d_unique_out, d_counts_out, d_num_runs_out, total_count_h));

    mgpu::scan(d_counts_out, matA.mRows, d_counts_out2, context);

    hipEventRecord(stop_rle_evt, 0);
    hipEventSynchronize(stop_rle_evt);

    hipEventElapsedTime(&elapsed, start_evt, stop_shuffle_evt);
    std::cout << "Shuffle elapsed: " << elapsed << std::endl;
    hipEventElapsedTime(&elapsed, start_evt, stop_carries_evt);
    std::cout << "Carries elapsed: " << elapsed << std::endl;
    hipEventElapsedTime(&elapsed, start_evt, stop_rle_evt);
    std::cout << "RLE/Total elapsed: " << elapsed << std::endl;

    /*
    CheckCuda(hipDeviceSynchronize());
    auto finish = std::chrono::system_clock::now();
    double elapsed = std::chrono::duration<double, std::milli>(finish - start).count();
    std::cout << "elapsed: " << elapsed << std::endl;
    */

    int h_num_runs_out;
    hipMemcpy(&h_num_runs_out, d_num_runs_out, sizeof(int), hipMemcpyDeviceToHost);
    std::cout << "h_num_runs_out = " << h_num_runs_out << std::endl;
    std::cout << "Readback and writing row pointers ..." << std::endl;
    hipMemcpy(h_row_ptrs, d_counts_out2, sizeof(int) * matA.mRows, hipMemcpyDeviceToHost);
    std::ofstream ofs("row_ptrs.bin");
    ofs.write((const char*)h_row_ptrs, sizeof(int) * h_num_runs_out);

    std::cout << "Readback and writing column indices ..." << std::endl;
    std::vector<uint32_t> res_columns = mgpu::from_mem(d_final_keys_cols);
    ofs.close();
    ofs.open("col_indices.bin");
    ofs.write((const char*)res_columns.data(), sizeof(uint32_t) * total_count_h);

    std::cout << "Readback and writing values ..." << std::endl;
    std::vector<float> res_values = mgpu::from_mem(d_final_vals);
    ofs.close();
    ofs.open("values.bin");
    ofs.write((const char*)res_values.data(), sizeof(float) * total_count_h);
}
