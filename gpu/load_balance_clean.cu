#include "hip/hip_runtime.h"
#include <moderngpu/cta_load_balance.hxx>
#include <moderngpu/cta_scan.hxx>
#include <moderngpu/cta_mergesort.hxx>

#include <hipcub/hipcub.hpp>

#include <iostream>
#include <fstream>
#include <random>
#include <cstring>
#include <chrono>
#include <map>
#include <unordered_map>

#include "MatrixMarket.h"
#include "DeviceMatrix.h"

#include "../MyHash.h"


// must match BLOCK_SIZE used to generate blocks in ../load-balance-test.cpp
#define BLOCK_SIZE          2048
#define ITEMS_PER_THREAD    16
#define NUM_THREADS         128

// TODO: Put this in a proper header file to share
class TSplit
{
public:
    int a_row;
    int bp;
    int b_col;
};


struct CustomLess
{
    template <typename DataType>
    __device__ bool operator()(const DataType& lhs, const DataType& rhs) { return lhs < rhs; }
};


__global__ void cuda_load_block_coop(const int *AmRowPtrs, const int *AmColIdx, const float *AmCSRVals,
                                     const int *BmRowPtrs, const int *BmColIdx, const float *BmCSRVals,
                                     const int *lb_data, const int *lb_block_ptrs, const TSplit* lb_thread_splits, float *output,
                                     int *out_keys, float *out_vals, int *atomic_p)
{
    int block = blockIdx.x;
    int cur_block_ptr = lb_block_ptrs[block];
    int next_block_ptr = lb_block_ptrs[block+1];
    int end_row = lb_data[next_block_ptr];
    int start_row = lb_data[cur_block_ptr];

    typedef hipcub::BlockRadixSort<int, NUM_THREADS, ITEMS_PER_THREAD, float, 4, true> BlockRadixSort;
    typedef hipcub::BlockReduce<float, NUM_THREADS> BlockReduce;
    __shared__ union {
        typename BlockRadixSort::TempStorage block_radix_storage;
        typename BlockReduce::TempStorage block_reduce_storage;
        float all_vals[BLOCK_SIZE];
    } smem;
    int thread_keys[ITEMS_PER_THREAD];
    float thread_vals[ITEMS_PER_THREAD];

    TSplit split = lb_thread_splits[block * NUM_THREADS + threadIdx.x];
    int row_end = AmRowPtrs[split.a_row + 1];
    int brow = AmColIdx[split.bp] - 1;
    int seg_end = BmRowPtrs[brow+1];

    int coeff_start = AmRowPtrs[split.a_row];
    if (split.a_row == end_row)
        seg_end = BmRowPtrs[brow] + lb_data[next_block_ptr + 2 + (split.bp - coeff_start)];
    float Acoeff = AmCSRVals[split.bp];

    for (int i = 0; i < ITEMS_PER_THREAD; i++)
    {
        if (split.b_col >= seg_end)
        {
            split.bp++;
            if (split.bp >= row_end) {
                split.a_row++;
                split.bp = AmRowPtrs[split.a_row];
                row_end = AmRowPtrs[split.a_row + 1];
            }
            Acoeff = AmCSRVals[split.bp];
            brow = AmColIdx[split.bp] - 1;
            split.b_col = BmRowPtrs[brow];
            seg_end = BmRowPtrs[brow+1];
            if (split.a_row == end_row) {
                int coeff_start = AmRowPtrs[split.a_row];
                seg_end = BmRowPtrs[brow] + lb_data[next_block_ptr + 2 + (split.bp - coeff_start)];
            }
        }

        thread_keys[i] = ((split.a_row - start_row) << 25) | BmColIdx[split.b_col];
        thread_vals[i] = BmCSRVals[split.b_col] * Acoeff;
        split.b_col++;
    }

    __syncthreads();

    BlockRadixSort(smem.block_radix_storage).Sort(thread_keys, thread_vals); //, 0, 24);

    __syncthreads();




/************************************************
 * Reduction phase
 */
    #define FULL_MASK   0xffffffff
    int warp = threadIdx.x / 32;
    int lane = threadIdx.x & 31;

    typedef hipcub::BlockLoad<int, NUM_THREADS, ITEMS_PER_THREAD> BlockLoadInt;
    typedef hipcub::BlockLoad<float, NUM_THREADS, ITEMS_PER_THREAD> BlockLoadFloat;
    typedef hipcub::BlockStore<int, NUM_THREADS, ITEMS_PER_THREAD> BlockStoreInt;
    typedef hipcub::BlockStore<float, NUM_THREADS, ITEMS_PER_THREAD> BlockStoreFloat;

    // XXX: Hard-coded 4 warps
    // Could combine these in the union below ...?
	__shared__ int delta_shared[4 + NUM_THREADS];	
    __shared__ float carry_out[NUM_THREADS];
    __shared__ int block_keys[4 + 1];       // reuse delta_shared?

    if (lane == 0)
        block_keys[warp] = thread_keys[0];

    int p = 0;
    float pcarry_out = 0;
    for (int i=1; i < ITEMS_PER_THREAD; i++)
    {
        if (thread_keys[p] == thread_keys[i]) {
            thread_vals[p] += thread_vals[i];
            pcarry_out = thread_vals[p];
        }
        else {
            p++;
            thread_keys[p] = thread_keys[i];
            thread_vals[p] = thread_vals[i];
            pcarry_out = 0;
        }
    }
    carry_out[threadIdx.x] = pcarry_out;

    int neighbor_key = __shfl_down_sync(FULL_MASK, thread_keys[0], 1);
    if (lane == 31) {
        neighbor_key = block_keys[warp+1];
    }
    // what happens to the very last warp carry_out?
    bool rseg_end = (p > 0);
    if (thread_keys[ITEMS_PER_THREAD-1] != neighbor_key) {
        p++;
        rseg_end = true;
        carry_out[threadIdx.x] = 0.0;
    }

    uint warp_mask = FULL_MASK >> (31 - lane);
    uint cta_mask = 0x7fffffff >> (31 - lane);

    uint warp_bits = __ballot_sync(FULL_MASK, rseg_end);
    delta_shared[warp] = warp_bits;
    __syncthreads();

    const int NumWarps = NUM_THREADS / 32;
    if (threadIdx.x < NumWarps) {
        uint cta_bits = __ballot_sync(0xf, 0 != delta_shared[threadIdx.x]);
        int warpSegment = 31 - __clz(cta_mask & cta_bits);
        int start = (-1 != warpSegment) ? 
            (31 - __clz(delta_shared[warpSegment]) + 32 * warpSegment) : 0;
        delta_shared[NumWarps + threadIdx.x] = start;
    }
    __syncthreads();
 
    // Find the closest flag to the left of this thread within the warp.
    // Include the flag for this thread.
    int start = 31 - __clz(warp_mask & warp_bits);
    if(-1 != start) start += ~31 & threadIdx.x;
    else start = delta_shared[NumWarps + warp];
    __syncthreads();
 
    uint tid_delta = threadIdx.x - start;

    for (int offset = 1; offset < NUM_THREADS; offset += offset)
    {
        if (tid_delta >= offset)
            carry_out[threadIdx.x] += carry_out[threadIdx.x - offset];
        __syncthreads();
    }

    // apply carry_out ...
    if (p > 0 && threadIdx.x > 0)
        thread_vals[0] += carry_out[threadIdx.x - 1];

    // do need inclusive sum 
    typedef hipcub::BlockScan<int, NUM_THREADS> BlockScanWriteOut;
    __shared__ typename BlockScanWriteOut::TempStorage bswo_storage;

    int start_p, total_p;
    BlockScanWriteOut(bswo_storage).ExclusiveSum(p, start_p, total_p);

    // TODO: coop write worth it?
    __shared__ int shared_p;
    if (threadIdx.x == 0)
        shared_p = atomicAdd(atomic_p, total_p);
    __syncthreads();

    start_p += shared_p;
    for (int i=0; i < p; i++) {
        out_keys[start_p + i] = thread_keys[i];
        out_vals[start_p + i] = thread_vals[i];
    }
//    out_counts[threadIdx.x] = p;


/*
 * End reduction phase
 **************************************************************/
}


int main(int argc, char **argv)
{
    mgpu::standard_context_t context;

    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " <mtx file>" << std::endl;
        exit(1);
    }

    MatrixMarket matA { argv[1] };
    MatrixMarket& matB = matA;
    DeviceMatrix dmA(matA, context);
    DeviceMatrix& dmB = dmA;

    std::ifstream if_data("../lb_data.bin", std::ifstream::binary);
    std::ifstream if_block_ptr("../lb_block_ptrs.bin", std::ifstream::binary);
    std::ifstream if_thread_splits("../lb_thread_splits.bin", std::ifstream::binary);

    if_data.seekg(0, if_data.end);
    if_block_ptr.seekg(0, if_block_ptr.end);
    if_thread_splits.seekg(0, if_thread_splits.end);
    int lb_data_size = if_data.tellg();
    int lb_block_ptrs_size = if_block_ptr.tellg();
    int lb_thread_splits_size = if_thread_splits.tellg();
    if_data.seekg(0, if_data.beg);
    if_block_ptr.seekg(0, if_block_ptr.beg);
    if_thread_splits.seekg(0, if_thread_splits.beg);

    std::cout << "lb_data size = " << lb_data_size << std::endl;
    std::cout << "lb_block_ptr size = " << lb_block_ptrs_size << std::endl;
    std::cout << "lb_thread_splits size = " << lb_thread_splits_size << std::endl;

    std::vector<int> lb_data, lb_block_ptrs;
    std::vector<std::vector<TSplit>> lb_thread_splits;
    lb_data.resize(lb_data_size >> 2);
    lb_block_ptrs.resize(lb_block_ptrs_size >> 2);
    lb_thread_splits.resize(lb_block_ptrs_size >> 2);
    for (int i=0; i < lb_thread_splits.size()-1; i++) {
        lb_thread_splits[i].resize(128);
        if_thread_splits.read((char *)lb_thread_splits[i].data(), sizeof(TSplit) * 128);
    }

    if_data.read((char *)lb_data.data(), lb_data_size);
    if_block_ptr.read((char *)lb_block_ptrs.data(), lb_block_ptrs_size);

    std::cout << "read lb_data = " << lb_data.size() << std::endl;
    std::cout << "read lb_block_ptrs = " << lb_block_ptrs.size() << std::endl;
    std::cout << "read lb_thread_splits" << std::endl;

    /*
    for (int i=1; i < lb_block_ptrs.size(); i++) {
        if (lb_data[lb_block_ptrs[i]] == lb_data[lb_block_ptrs[i-1]]) {
            std::cout << "single row: " << i << std::endl;
            break;
        }
    }
    */

    std::vector<TSplit> lb_flat_splits;
    for (int i = 0; i < lb_thread_splits.size()-1; i++)
    {
        for (int j = 0; j < NUM_THREADS; j++)
            lb_flat_splits.push_back(lb_thread_splits[i][j]);
    }
    mgpu::mem_t<int> d_lb_data = mgpu::to_mem(lb_data, context);
    mgpu::mem_t<int> d_lb_block_ptrs = mgpu::to_mem(lb_block_ptrs, context);
    mgpu::mem_t<TSplit> d_flat_tsplits = mgpu::to_mem(lb_flat_splits, context);

    std::vector<float> out_buffer;
    std::vector<int> out_keys;
    std::vector<float> out_vals;
    out_keys.resize(100000000);
    out_vals.resize(100000000);
    std::vector<int> atomic_p;
    atomic_p.resize(2);

    out_buffer.resize(lb_block_ptrs.size());
    mgpu::mem_t<float> d_output = mgpu::to_mem(out_buffer, context);
    mgpu::mem_t<int> d_out_keys = mgpu::to_mem(out_keys, context);
    mgpu::mem_t<float> d_out_vals = mgpu::to_mem(out_vals, context);
    mgpu::mem_t<int> d_atomic_p = mgpu::to_mem(atomic_p, context);

    /*
    int numBlocks;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks, cuda_load_block_coop, NUM_THREADS, 2048 * 4);
    std::cout << "max occupancy(?) = " << numBlocks << std::endl;
    */

    auto start = std::chrono::system_clock::now();
    cuda_load_block_coop<<<lb_block_ptrs.size()-1, NUM_THREADS>>>(dmA.raw.d_row_ptrs, dmA.raw.d_col_idx, dmA.raw.d_values,
                                             dmB.raw.d_row_ptrs, dmB.raw.d_col_idx, dmB.raw.d_values,
                                             d_lb_data.data(), d_lb_block_ptrs.data(), d_flat_tsplits.data(), d_output.data(), d_out_keys.data(), d_out_vals.data(), d_atomic_p.data());
    hipDeviceSynchronize();
    auto finish = std::chrono::system_clock::now();
    double elapsed = std::chrono::duration<double, std::milli>(finish - start).count();
    std::cout << "elapsed: " << elapsed << std::endl;

    std::vector<float> read_back = mgpu::from_mem(d_output);
    std::cout << "Result 0: " << read_back[0] << std::endl;

    return 0;
}
