#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include "MatrixMarket.h"
#include "DeviceMatrix.h"

#include <moderngpu/kernel_segreduce.hxx>
#include <moderngpu/kernel_scan.hxx>
#include <moderngpu/kernel_compact.hxx>

#include <hipcub/hipcub.hpp>


constexpr int BLOCK_SIZE = 2048;

constexpr int NUM_ITERS = 1;
constexpr int NUM_THREADS_SPLIT = 6;
constexpr int NUM_THREADS_SCAN_GEN = 128;

void CheckCuda(hipError_t success)
{
    if (success != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(success) << std::endl;
        exit(1);
    }
}


constexpr int MAX_ELEMENT = 999999999;

struct node_t {
    int value;
    int list;
};


__device__ void tournament_tree_kth_largest(int **A, int *b, int m, int w_k, int k, node_t *T, int SIZE)
{
    int POT = (32 - __clz(SIZE)) - 1;
    for (int i=0; i < m; i++) {
        if (b[i] == 0) {
            T[SIZE + i].value = -MAX_ELEMENT;
            T[SIZE + i].list = -1;
            continue;
        }
        T[SIZE + i].value = A[i][b[i]-1] - 1;
        T[SIZE + i].list = i;
    }

    // First iteration, just propagate up the tree
    for (int l = POT-1; l >= 0; l--)
    {
        int l0 = 1 << l;
        int l1 = 1 << (l+1);
        for (int j = 0; j < l0; j++)
        {
            if (T[l1 + j*2].value > T[l1 + j*2 + 1].value)
                T[l0 + j] = T[l1 + j*2];
            else
                T[l0 + j] = T[l1 + j*2+1];
        }
    }

    int winner = T[1].list;
    b[winner] -= w_k;

    T[SIZE + winner].list = winner;
    if (b[winner] == 0)
        T[SIZE + winner].value = -MAX_ELEMENT;
    else
        T[SIZE + winner].value = A[winner][b[winner]-1] - 1;

    // Now just propagate the winning list
    for (int i = 0; i < k-1; i++)
    {
        int j = winner;
        for (int l = POT-1; l >= 0; l--)
        {
            int l0 = 1 << l;
            int l1 = 1 << (l+1);
            int j_floor = (j >> 1) * 2;
            if (T[l1 + j_floor].value > T[l1 + j_floor + 1].value)
                T[l0 + (j >> 1)] = T[l1 + j_floor];
            else
                T[l0 + (j >> 1)] = T[l1 + j_floor + 1];
            j = j >> 1;
        }
        winner = T[1].list;
        b[winner] -= w_k;
        
        T[SIZE + winner].list = winner;
        if (b[winner] == 0)
            T[SIZE + winner].value = -MAX_ELEMENT;
        else
            T[SIZE + winner].value = A[winner][b[winner]-1] - 1;
   }
}


__device__ void tournament_tree_kth_largest_reverse(int **A, int *alen, int *b, int m, int w_k, int k, node_t *T, int SIZE)
{
    int POT = (32 - __clz(SIZE)) - 1;
    for (int i=0; i < m; i++) {
        if (b[i] == alen[i]) {
            T[SIZE + i].value = -MAX_ELEMENT;
            T[SIZE + i].list = -1;
            continue;
        }
        T[SIZE + i].value = -A[i][b[i]] + 1;
        T[SIZE + i].list = i;
    }

    for (int i=m; i < SIZE; i++) {
        T[SIZE + i].value = -MAX_ELEMENT;
        T[SIZE + i].list = -1;
    }

    // First iteration, just propagate up the tree
    for (int l = POT-1; l >= 0; l--)
    {
        int l0 = 1 << l;
        int l1 = 1 << (l+1);
        for (int j = 0; j < l0; j++)
        {
            if (T[l1 + j*2].value > T[l1 + j*2 + 1].value)
                T[l0 + j] = T[l1 + j*2];
            else
                T[l0 + j] = T[l1 + j*2+1];
        }
    }

    int winner = T[1].list;
    b[winner] += w_k;

    T[SIZE + winner].list = winner;
    if (b[winner] >= alen[winner])
        T[SIZE + winner].value = -MAX_ELEMENT;
    else
        T[SIZE + winner].value = -A[winner][b[winner]] + 1;

    // Now just propagate the winning list
    for (int i = 0; i < k-1; i++)
    {
        int j = winner;
        for (int l = POT-1; l >= 0; l--)
        {
            int l0 = 1 << l;
            int l1 = 1 << (l+1);
            int j_floor = (j >> 1) * 2;
            if (T[l1 + j_floor].value > T[l1 + j_floor + 1].value)
                T[l0 + (j >> 1)] = T[l1 + j_floor];
            else
                T[l0 + (j >> 1)] = T[l1 + j_floor + 1];
            j = j >> 1;
        }
        winner = T[1].list;
        b[winner] += w_k;

        T[SIZE + winner].list = winner;
        if (b[winner] >= alen[winner])
            T[SIZE + winner].value = -MAX_ELEMENT;
        else
            T[SIZE + winner].value = -A[winner][b[winner]] + 1;
   }
}


__device__ void tournament_tree_kth_smallest(int **A, int *alen, int *b, int m, int w_k, int k, node_t *T, int SIZE)
{
    int POT = (32 - __clz(SIZE)) - 1;
    for (int i=0; i < m; i++) {
        if (b[i] + w_k > alen[i]) {
            T[SIZE + i].value = MAX_ELEMENT;
            T[SIZE + i].list = -1;
            continue;
        }
        T[SIZE + i].value = A[i][b[i] + w_k - 1] - 1;
        T[SIZE + i].list = i;
    }

    // First iteration, just propagate up the tree
    for (int l = POT-1; l >= 0; l--)
    {
        int l0 = (1 << l);
        int l1 = (1 << (l+1));
        for (int j = 0; j < (1 << l); j++)
        {
            if (T[l1 + j*2].value < T[l1 + j*2 + 1].value)
                T[l0 + j] = T[l1 + j*2];
            else
                T[l0 + j] = T[l1 + j*2+1];
        }
    }

    int winner = T[1].list;
    b[winner] += w_k;

    T[SIZE + winner].list = winner;
    if (b[winner] + w_k > alen[winner])
        T[SIZE + winner].value = MAX_ELEMENT;
    else
        T[SIZE + winner].value = A[winner][b[winner]+w_k-1] - 1;

    // Now just propagate the winning list
    for (int i = 0; i < k-1; i++)
    {
        int j = winner;
        for (int l = POT-1; l >= 0; l--)
        {
            int l0 = 1 << l;
            int l1 = 1 << (l+1);
            int j_floor = (j >> 1) * 2;
            if (T[l1 + j_floor].value < T[l1 + j_floor + 1].value)
                T[l0 + (j >> 1)] = T[l1 + j_floor];
            else
                T[l0 + (j >> 1)] = T[l1 + j_floor + 1];
            j = j >> 1;
        }
        winner = T[1].list;
        b[winner] += w_k;

        T[SIZE + winner].list = winner;
        if (b[winner] + w_k > alen[winner])
            T[SIZE + winner].value = MAX_ELEMENT;
        else
            T[SIZE + winner].value = A[winner][b[winner]+w_k-1] - 1;
   }
}


__device__ void tournament_tree_kth_smallest_reverse(int **A, int *alen, int *b, int m, int w_k, int k, node_t *T, int SIZE)
{
    int POT = (32 - __clz(SIZE)) - 1;
    for (int i=0; i < m; i++) {
        if (b[i] - w_k < 0) {
            T[SIZE + i].value = MAX_ELEMENT;
            T[SIZE + i].list = -1;
            continue;
        }
        T[SIZE + i].value = -A[i][b[i] - w_k] + 1;
        T[SIZE + i].list = i;
    }

    for (int i=m; i < SIZE; i++) {
        T[SIZE + i].value = MAX_ELEMENT;
        T[SIZE + i].list = -1;
    }

    // First iteration, just propagate up the tree
    for (int l = POT-1; l >= 0; l--)
    {
        int l0 = (1 << l);
        int l1 = (1 << (l+1));
        for (int j = 0; j < (1 << l); j++)
        {
            if (T[l1 + j*2].value < T[l1 + j*2 + 1].value)
                T[l0 + j] = T[l1 + j*2];
            else
                T[l0 + j] = T[l1 + j*2+1];
        }
    }

    int winner = T[1].list;
    b[winner] -= w_k;

    T[SIZE + winner].list = winner;
    if (b[winner] - w_k < 0)
        T[SIZE + winner].value = MAX_ELEMENT;
    else
        T[SIZE + winner].value = -A[winner][b[winner]-w_k] + 1;

    // Now just propagate the winning list
    for (int i = 0; i < k-1; i++)
    {
        int j = winner;
        for (int l = POT-1; l >= 0; l--)
        {
            int l0 = 1 << l;
            int l1 = 1 << (l+1);
            int j_floor = (j >> 1) * 2;
            if (T[l1 + j_floor].value < T[l1 + j_floor + 1].value)
                T[l0 + (j >> 1)] = T[l1 + j_floor];
            else
                T[l0 + (j >> 1)] = T[l1 + j_floor + 1];
            j = j >> 1;
        }
        winner = T[1].list;
        b[winner] -= w_k;

        T[SIZE + winner].list = winner;
        if (b[winner] - w_k < 0)
            T[SIZE + winner].value = MAX_ELEMENT;
        else
            T[SIZE + winner].value = -A[winner][b[winner]-w_k] + 1;
   }
}


// +/- 1 is because we are zero-based while the Python reference code is 1-based using scipy
__device__ inline int compute_lmax(int **A, int *b, int blen)
{
    int lmax = -MAX_ELEMENT;
    for (int i=0; i < blen; i++) {
        if (b[i] > 0 && (A[i][b[i]-1]-1) > lmax)
            lmax = A[i][b[i]-1] - 1;
    }
    return lmax;
}

__device__ inline int compute_lmax_reverse(int **A, int *alen, int *b, int blen)
{
    int lmax = -MAX_ELEMENT;
    for (int i=0; i < blen; i++) {
        if (b[i] < alen[i] && (-A[i][b[i]]+1) > lmax)
            lmax = -A[i][b[i]]+1;
    }
    return lmax;
}

__device__ inline bool compute_carry(int **A, int *alen, int *b, int blen, int lmax)
{
    for (int i=0; i < blen; i++)
    {
        if (b[i] < alen[i] && A[i][b[i]]-1 == lmax)
            return true;
    }
    return false;
}

__device__ inline bool compute_carry_reverse(int **A, int *alen, int *b, int blen, int lmax)
{
    for (int i=0; i < blen; i++)
    {
        if (b[i] > 0 && -A[i][b[i]-1]+1 == lmax)
            return true;
    }
    return false;
}


__device__ bool row_splitter(int **A, int *alen, int *b, int m, int p, node_t *T, int MAXSIZE)
{
    // assert m < MAXSIZE
    int n_max = -1;
    for (int i=0; i < m; i++) {
        b[i] = 0;
        n_max = (alen[i] > n_max) ? alen[i] : n_max;
    }

    if (p == 0)
        return false;

    // Do this fill in once instead of every call to TT
    // kth_smallest
    for (int i = 0; i < MAXSIZE; i++)
    {
        T[MAXSIZE + i].value = MAX_ELEMENT;
        T[MAXSIZE + i].list = -1;
    }

    // kth_largest
    for (int i = 0; i < MAXSIZE; i++)
    {
        T[MAXSIZE*3 + i].value = -MAX_ELEMENT;
        T[MAXSIZE*3 + i].list = -1;
    }

    // Handle short splits
    if (p < m) {
        tournament_tree_kth_smallest(A, alen, b, m, 1, p, T, MAXSIZE);
        int lmax = compute_lmax(A, b, m);
        return compute_carry(A, alen, b, m, lmax);
    }

    int r = ceilf(logf((float)p / m) / logf(2.0));
    int two_r = 1 << r;
    int alpha = n_max / two_r;              // implicit floor
    int n = two_r * (alpha + 1) - 1;
    int k = ceilf((float)p / n * alpha);

    // Initial partition for the recursion
    tournament_tree_kth_smallest(A, alen, b, m, two_r, k, T, MAXSIZE);
    int lmax = compute_lmax(A, b, m);

    if (lmax == MAX_ELEMENT)
    {
        // XXX: Need to handle this case
        return;
    }

    // r iterative steps
    for (int k=0; k < r; k++)
    {
        int Lsize = 0;
        int w_k = 1 << (r - k - 1);
        int target_size = ceilf((float)p * (n / w_k) / n);

        for (int i=0; i < m; i++)
            Lsize += b[i] / w_k;

        for (int i=0; i < m; i++)
        {
            if (b[i] + w_k > alen[i])
                continue;
            int undecided = A[i][b[i] + w_k - 1] - 1;
            if (undecided < lmax) {
                b[i] += w_k;
                Lsize++;
            }
        }
        if (Lsize > target_size) {
            tournament_tree_kth_largest(A, b, m, w_k, Lsize - target_size, T + (MAXSIZE*2), MAXSIZE);
        }
        if (Lsize < target_size) {
            tournament_tree_kth_smallest(A, alen, b, m, w_k, target_size - Lsize, T, MAXSIZE);
        }

        lmax = compute_lmax(A, b, m);
    }

    return compute_carry(A, alen, b, m, lmax);
}


__device__ bool row_splitter_reverse(int **A, int *alen, int *b, int m, int p, node_t *T, int MAXSIZE)
{
    int n_max = -1;
    for (int i=0; i < m; i++) {
        b[i] = alen[i];
        n_max = (alen[i] > n_max) ? alen[i] : n_max;
    }

    if (p == 0)
        return false;

    // Handle short splits
    if (p < m) {
        tournament_tree_kth_smallest_reverse(A, alen, b, m, 1, p, T, MAXSIZE);
        int lmax = compute_lmax_reverse(A, alen, b, m);
        return compute_carry_reverse(A, alen, b, m, lmax);
    }

    int r = ceilf(logf((float)p / m) / logf(2.0));
    int two_r = 1 << r;
    int alpha = n_max / two_r;              // implicit floor
    int n = two_r * (alpha + 1) - 1;
    int k = ceilf((float)p / n * alpha);
    
    // Initial partition for the recursion
    tournament_tree_kth_smallest_reverse(A, alen, b, m, two_r, k, T, MAXSIZE);
    int lmax = compute_lmax_reverse(A, alen, b, m);

    // r iterative steps
    for (int k=0; k < r; k++)
    {
        int Lsize = 0;
        int w_k = 1 << (r - k - 1);
        int target_size = ceilf((float)p * (n / w_k) / n);

        for (int i=0; i < m; i++)
            Lsize += (alen[i] - b[i]) / w_k;

        for (int i=0; i < m; i++)
        {
            if (b[i] - w_k < 0)
                continue;
            int undecided = -A[i][b[i] - w_k] + 1;
            if (undecided < lmax) {
                b[i] -= w_k;
                Lsize++;
            }
        }
        if (Lsize > target_size) {
            tournament_tree_kth_largest_reverse(A, alen, b, m, w_k, Lsize - target_size, T + (MAXSIZE * 2), MAXSIZE);
        }
        if (Lsize < target_size) {
            tournament_tree_kth_smallest_reverse(A, alen, b, m, w_k, target_size - Lsize, T, MAXSIZE);
        }

        lmax = compute_lmax_reverse(A, alen, b, m);
    }

    return compute_carry_reverse(A, alen, b, m, lmax);
}


struct block_data_t {
    int row;
    int p;
    bool reverse;
};

__global__ void split_matrix_fwd(int *row_ptrs, int *col_idx, int *Brow_ptrs, int *Bcol_idx, int *base, bool *carry_out, block_data_t *splits, int *out_ptrs, int *work_ptrs, int *workspace, int nsplits, int *indices)
{
    int threadId = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (threadId >= nsplits)
        return;

    threadId = indices[threadId];
    int row = splits[threadId].row;
    int m = row_ptrs[row+1] - row_ptrs[row];
    int npot = 1 << (32 - __clz(m-1));
    int p = splits[threadId].p;
    int *b = base + out_ptrs[threadId];

    uintptr_t Aaddr = (uintptr_t)(workspace + work_ptrs[threadId]);
    if (Aaddr % 8 != 0)
        Aaddr += 8 - (Aaddr % 8);
    int **A = (int **)Aaddr;
    Aaddr += m * 8;
    int *alen = (int *)Aaddr;
    Aaddr += m * 4;
    node_t *T = (node_t *)Aaddr;

    // assert m < MAXSIZE or handle exception
    for (int i=0; i < m; i++) {
        int brow = col_idx[row_ptrs[row] + i] - 1;
        A[i] = Bcol_idx + Brow_ptrs[brow];
        alen[i] = Brow_ptrs[brow+1] - Brow_ptrs[brow];
    }

    carry_out[threadId] = row_splitter(A, alen, b, m, p, T, npot);
}


__global__ void split_matrix_reverse(int *row_ptrs, int *col_idx, int *Brow_ptrs, int *Bcol_idx, int *base, bool *carry_out, block_data_t *splits, int *out_ptrs, int *work_ptrs, int *workspace, int nsplits, int *indices)
{
    int threadId = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (threadId >= nsplits)
        return;

    threadId = indices[threadId];
    int row = splits[threadId].row;
    int m = row_ptrs[row+1] - row_ptrs[row];
    int npot = 1 << (32 - __clz(m-1));
    int p = splits[threadId].p;
    int *b = base + out_ptrs[threadId];


    uintptr_t Aaddr = (uintptr_t)(workspace + work_ptrs[threadId]);
    if (Aaddr % 8 != 0)
        Aaddr += 8 - (Aaddr % 8);
    int **A = (int **)Aaddr;
    Aaddr += m * 8;
    int *alen = (int *)Aaddr;
    Aaddr += m * 4;
    node_t *T = (node_t *)Aaddr;

    // assert m < MAXSIZE or handle exception
    for (int i=0; i < m; i++) {
        int brow = col_idx[row_ptrs[row] + i] - 1;
        A[i] = Bcol_idx + Brow_ptrs[brow];
        alen[i] = Brow_ptrs[brow+1] - Brow_ptrs[brow];
    }

    carry_out[threadId] = row_splitter_reverse(A, alen, b, m, p, T, npot);
}


constexpr int ROWS_PER_THREAD = 14;
__global__ void scan_gen_blocks(int *cum_row_sizes, int *row_sizes, block_data_t *out, int nblocks, int nrows,
                                int *fwd_indices, int *reverse_indices, unsigned int *block_indices_cnt)
{
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    int start = tid * ROWS_PER_THREAD;
    if (start > nrows)
        return;

    int last_rsize = cum_row_sizes[start];
    int last_block = last_rsize / BLOCK_SIZE;
    for (int i=start+1; i <= start + ROWS_PER_THREAD; i++) {
        if (i > nrows)
            return;
        int row_size = cum_row_sizes[i];
        int block = row_size / BLOCK_SIZE;
        while (block > last_block) {
            out[last_block].row = i;
            out[last_block].p = ((last_block+1) * BLOCK_SIZE) - last_rsize;
            if (out[last_block].p / (float)row_sizes[i] > 0.5) {
                out[last_block].p = row_sizes[i] - out[last_block].p;
                out[last_block].reverse = true;
                reverse_indices[atomicAdd(block_indices_cnt + 1, 1)] = last_block;
            }
            else {
                out[last_block].reverse = false;
                fwd_indices[atomicAdd(block_indices_cnt, 1)] = last_block;
            }
            last_block++;
        }
        last_rsize = row_size;
    }
}


__global__ void compute_partial_row_sizes(int *Arowptr, int *Acolidx, int *Browlens, int *out_sizes, int nrows)
{
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (tid > nrows)
        return;

    int size = 0;
    // tid is the row we're computing partial sizes for
    for (int i=Arowptr[tid]; i < Arowptr[tid+1]; i++)
    {
        int brow = Acolidx[i] - 1;
        size += Browlens[brow];
    }

    out_sizes[tid] = size;
}


struct SelectReverse
{
    __host__ __device__ __forceinline__
    SelectReverse() {}

    __host__ __device__ __forceinline__
    bool operator()(const block_data_t& a) const {
        return a.reverse;
    }
};
struct SelectForward
{
    __host__ __device__ __forceinline__
    SelectForward() {}

    __host__ __device__ __forceinline__
    bool operator()(const block_data_t& a) const {
        return !a.reverse;
    }
};


int main(int argc, char **argv)
{
    mgpu::standard_context_t context;

    if (argc < 3) {
        std::cerr << "Usage: " << argv[0] << " <mtx file> <mtx file>" << std::endl;
        exit(1);
    }

    // Initial setup stuff
    MatrixMarket matA { argv[1] };
    MatrixMarket matB { argv[2] };
    DeviceMatrix dmA(matA, context);
    DeviceMatrix dmB(matB, context);

    float time;
    hipEvent_t start, finish_psizes, finish_gen_splits, finish_fwd, stop;
    CheckCuda(hipEventCreate(&start));
    CheckCuda(hipEventCreate(&finish_psizes));
    CheckCuda(hipEventCreate(&finish_gen_splits));
    CheckCuda(hipEventCreate(&finish_fwd));
    CheckCuda(hipEventCreate(&stop));

    mgpu::mem_t<int> d_row_sizes(matA.mRows, context);
    mgpu::mem_t<int> d_cumrow_sizes(matA.mRows, context);
    mgpu::mem_t<int> d_total_partials(1, context);

    int scan_blocks = (matA.mRows / (ROWS_PER_THREAD * NUM_THREADS_SCAN_GEN)) + 1;
    mgpu::mem_t<int> d_out_final(1, context);
    mgpu::mem_t<int> d_work_final(1, context);

    mgpu::mem_t<int> Browlens(matA.mRows, context);     // should be matB

    hipDeviceSynchronize();

    // Find the rows and where to split them (on GPU)
    hipEventRecord(start, 0);
    int *Acolidx = dmA.raw.d_col_idx;
    int *Arowptrs = dmA.raw.d_row_ptrs;
    int *Browptrs = dmB.raw.d_row_ptrs;
    int *d_Browlens = Browlens.data(); 
    mgpu::transform([=]MGPU_DEVICE(int index)
        {
            d_Browlens[index] = Browptrs[index+1] - Browptrs[index];
        }, matA.mRows, context);        // should be matB.mRows

    int n_partials_blocks = (matA.mRows / 128) + 1;
    std::cout << "n_partials_blocks = " << n_partials_blocks << std::endl;
    compute_partial_row_sizes<<<n_partials_blocks, 128>>>(Arowptrs, Acolidx, d_Browlens, d_row_sizes.data(), matA.mRows);

    mgpu::scan<mgpu::scan_type_inc>(d_row_sizes.data(), matA.mRows, d_cumrow_sizes.data(), mgpu::plus_t<int>(),
                                    d_total_partials.data(), context);
    hipEventRecord(finish_psizes, 0);

    int total_partials = mgpu::from_mem(d_total_partials)[0];
    int total_blocks = total_partials / BLOCK_SIZE;
    std::cout << "Num blocks = " << total_blocks << std::endl;

    // have scan_gen_blocks split the indices into forward and reverse arrays using atomics
    mgpu::mem_t<int> fwd_block_indices(total_blocks, context);
    mgpu::mem_t<int> reverse_block_indices(total_blocks, context);
    std::vector<unsigned int> init_block_indices_cnt = { 0, 0 };
    mgpu::mem_t<unsigned int> block_indices_cnt = mgpu::to_mem(init_block_indices_cnt, context);

    mgpu::mem_t<bool> d_carry_out(total_blocks+1, context);
    mgpu::mem_t<int> d_out_ptrs(total_blocks, context);
    mgpu::mem_t<int> d_work_ptrs(total_blocks, context);
    mgpu::mem_t<block_data_t> block_data(total_blocks, context);
    hipMemset(block_data.data(), 0, sizeof(block_data_t) * total_blocks);
    std::cout << "scan_blocks = " << scan_blocks << ", total_blocks = " << total_blocks << std::endl;
    scan_gen_blocks<<<scan_blocks, NUM_THREADS_SCAN_GEN>>>(d_cumrow_sizes.data(), d_row_sizes.data(), block_data.data(), total_blocks, matA.mRows,
                                                           fwd_block_indices.data(), reverse_block_indices.data(),
                                                            block_indices_cnt.data());

    // prefix sum to get block 'out' value
    block_data_t *d_block_data = block_data.data();
    auto out_scan = [=]MGPU_DEVICE(int index)
        {
            int r = d_block_data[index].row;
            int m = Arowptrs[r + 1] - Arowptrs[r];
            return m;
        };
    mgpu::transform_scan<int>(out_scan, total_blocks, d_out_ptrs.data(), mgpu::plus_t<int>(), d_out_final.data(), context);

    // duplication with kernel above, but hopefully this isn't too slow
    mgpu::transform_scan<int>([=]MGPU_DEVICE(int index) {
        int r = d_block_data[index].row;
        int m = Arowptrs[r + 1] - Arowptrs[r];
        // XXX: Make sure m >= 1
        int npot = 1 << (32 - __clz(m - 1));
        // Fix this mess, A is a pointer so 2 words, alen is an array of words, so another m words + 16 for alignment
        // 2 * npot for node_t but node_ts are 8 bytes
        return 3 * m + 8 * npot + 16;
    }, total_blocks, d_work_ptrs.data(), mgpu::plus_t<int>(), d_work_final.data(), context);

    // Allocate working memory for the variable sizes
    int total_work_size = mgpu::from_mem(d_work_final)[0];
    std::cout << "Working memory size = " << total_work_size * sizeof(int) << std::endl;
    mgpu::mem_t<int> d_work(total_work_size * 2, context);

    hipEventRecord(finish_gen_splits, 0);

    std::vector<unsigned int> h_block_counters = mgpu::from_mem(block_indices_cnt);
    std::cout << "Num forward blocks: " << h_block_counters[0] << std::endl;
    std::cout << "Num reverse blocks: " << h_block_counters[1] << std::endl;

    int total_split_size = mgpu::from_mem(d_out_final)[0];
    mgpu::mem_t<int> d_output(total_split_size, context);

    // total_blocks is number of split blocks, while n_blocks is CUDA blocks to compute that many splits
    int n_blocks = (h_block_counters[0] / NUM_THREADS_SPLIT) + 1;
    std::cout << "Using " << n_blocks << " CUDA blocks for forward splits." << std::endl;
    split_matrix_fwd<<<n_blocks, NUM_THREADS_SPLIT>>>(dmA.raw.d_row_ptrs, dmA.raw.d_col_idx, dmB.raw.d_row_ptrs, dmB.raw.d_col_idx, d_output.data(),
                                            d_carry_out.data(), block_data.data(), d_out_ptrs.data(), d_work_ptrs.data(), d_work.data(), h_block_counters[0], fwd_block_indices.data());
    hipEventRecord(finish_fwd, 0);

    n_blocks = (h_block_counters[1] / NUM_THREADS_SPLIT) + 1;
    std::cout << "Using " << n_blocks << " CUDA blocks for reverse splits." << std::endl;
    split_matrix_reverse<<<n_blocks, NUM_THREADS_SPLIT>>>(dmA.raw.d_row_ptrs, dmA.raw.d_col_idx, dmB.raw.d_row_ptrs, dmB.raw.d_col_idx, d_output.data(),
                                            d_carry_out.data(), block_data.data(), d_out_ptrs.data(), d_work_ptrs.data(), d_work.data(), h_block_counters[1], reverse_block_indices.data());

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, finish_psizes);
    std::cout << "Finished partial sizes: " << time << " ms" << std::endl;
    hipEventElapsedTime(&time, start, finish_gen_splits);
    std::cout << "Finished finding split points: " << time << " ms" << std::endl;
    hipEventElapsedTime(&time, start, stop);
    std::cout << "Finished computing splits: " << time << " ms" << std::endl;

    hipDeviceSynchronize();
    CheckCuda(hipGetLastError());


    // Finally, read the data back and write it into two files: lb_block_ptrs.bin and lb_data.bin
    // in the format that load_balance_clean expects. Specifically:
    // lb_block_ptrs.bin is simply the indices of the blocks in lb_data.bin
    // lb_data.bin is an array of split data:
    //   word 0 = row
    //   word 1 = number of non-zeros in row == number of split points
    //            (not entirely necessary, as this can be easily computed from the row)
    //   word 2 = carry flag (whether we need to apply carries between this block and the next)
    //   word 3..k+3 = k split points for the row
    std::vector<int> h_out_ptrs = mgpu::from_mem(d_out_ptrs);
    std::ofstream lb_block_ptrs_file("lb_block_ptrs.bin");
    int d = 0;
    int first_row_size = matA.mRowPtrs[1] - matA.mRowPtrs[0];
    lb_block_ptrs_file.write((const char *)&d, sizeof(d));
    for (int i=0; i < total_blocks; i++)
    {
        d = (i * 3) + h_out_ptrs[i] + first_row_size + 3;
        lb_block_ptrs_file.write((const char *)&d, sizeof(d));
    }

    // Copy back block data from GPU (from_mem with bool array doesn't work with mgpu, so using hipMemcpy)
    bool *h_carry_out = new bool[total_blocks+1];
    CheckCuda(hipMemcpy(h_carry_out, d_carry_out.data(), sizeof(bool) * (total_blocks+1), hipMemcpyDeviceToHost));
    std::vector<int> h_output = mgpu::from_mem(d_output);
    std::vector<block_data_t> h_block_data = mgpu::from_mem(block_data);

    std::ofstream lb_data_file("lb_data.bin");
    /* first block is all 0 */
    d = 0;
    lb_data_file.write((const char *)&d, sizeof(d));     // first is the row ( == 0 )
    lb_data_file.write((const char *)&first_row_size, sizeof(first_row_size));      // next number of splits
    int bcarry = h_carry_out[0];
    lb_data_file.write((const char *)&bcarry, sizeof(int));        // next is the carry which is 0 for the first block
    // next is the split points. For the first row, these are all zero (and d is zero)
    for (int i=0; i < first_row_size; i++)
        lb_data_file.write((const char *)&d, sizeof(d));

    for (int block_idx=0; block_idx < total_blocks; block_idx++)
    {
        int row = h_block_data[block_idx].row;
        lb_data_file.write((const char *)&row, sizeof(row));
        int split_row_size = matA.mRowPtrs[row + 1] - matA.mRowPtrs[row];
        lb_data_file.write((const char *)&split_row_size, sizeof(split_row_size));
        bcarry = h_carry_out[block_idx+1];      // convert bool carry to int
        // last block is always 0 carry?
        if (block_idx == total_blocks - 1)
            lb_data_file.write((const char *)&d, sizeof(int));
        else
            lb_data_file.write((const char *)&bcarry, sizeof(int));
        lb_data_file.write((const char *)(h_output.data() + h_out_ptrs[block_idx]), sizeof(int) * split_row_size);
    }
}
