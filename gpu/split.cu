#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include "MatrixMarket.h"
#include "DeviceMatrix.h"

constexpr int NUM_ITERS = 10;
constexpr int NUM_THREADS = 64;

void CheckCuda(hipError_t success)
{
    if (success != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(success) << std::endl;
        exit(1);
    }
}


constexpr int MAX_ELEMENT = 999999999;

template <int POT=5>
__device__ void tournament_tree_kth_largest(int **A, int *b, int m, int w_k, int k)
{
    constexpr int SIZE = 1 << POT;
    struct node_t {
        int value;
        int list;
    } T[SIZE * 2];

    for (int i=0; i < m; i++) {
        if (b[i] == 0) {
            T[SIZE + i].value = -MAX_ELEMENT;
            T[SIZE + i].list = -1;
            continue;
        }
        T[SIZE + i].value = A[i][b[i]-1] - 1;
        T[SIZE + i].list = i;
    }

    for (int i=m; i < SIZE; i++) {
        T[SIZE + i].value = -MAX_ELEMENT;
        T[SIZE + i].list = -1;
    }

    // First iteration, just propagate up the tree
    for (int l = POT-1; l >= 0; l--)
    {
        int l0 = 1 << l;
        int l1 = 1 << (l+1);
        for (int j = 0; j < l0; j++)
        {
            if (T[l1 + j*2].value > T[l1 + j*2 + 1].value)
                T[l0 + j] = T[l1 + j*2];
            else
                T[l0 + j] = T[l1 + j*2+1];
        }
    }

    int winner = T[1].list;
    b[winner] -= w_k;

    T[SIZE + winner].list = winner;
    if (b[winner] == 0)
        T[SIZE + winner].value = -MAX_ELEMENT;
    else
        T[SIZE + winner].value = A[winner][b[winner]-1] - 1;

    // Now just propagate the winning list
    for (int i = 0; i < k-1; i++)
    {
        int j = winner;
        for (int l = POT-1; l >= 0; l--)
        {
            int l0 = 1 << l;
            int l1 = 1 << (l+1);
            int j_floor = (j >> 1) * 2;
            if (T[l1 + j_floor].value > T[l1 + j_floor + 1].value)
                T[l0 + (j >> 1)] = T[l1 + j_floor];
            else
                T[l0 + (j >> 1)] = T[l1 + j_floor + 1];
            j = j >> 1;
        }
        winner = T[1].list;
        b[winner] -= w_k;
        
        T[SIZE + winner].list = winner;
        if (b[winner] == 0)
            T[SIZE + winner].value = -MAX_ELEMENT;
        else
            T[SIZE + winner].value = A[winner][b[winner]-1] - 1;
   }
}


template <int POT=5>
__device__ void tournament_tree_kth_largest_reverse(int **A, int *alen, int *b, int m, int w_k, int k)
{
    constexpr int SIZE = 1 << POT;
    struct node_t {
        int value;
        int list;
    } T[SIZE * 2];

    for (int i=0; i < m; i++) {
        if (b[i] == alen[i]) {
            T[SIZE + i].value = -MAX_ELEMENT;
            T[SIZE + i].list = -1;
            continue;
        }
        T[SIZE + i].value = -A[i][b[i]] + 1;
        T[SIZE + i].list = i;
    }

    for (int i=m; i < SIZE; i++) {
        T[SIZE + i].value = -MAX_ELEMENT;
        T[SIZE + i].list = -1;
    }

    // First iteration, just propagate up the tree
    for (int l = POT-1; l >= 0; l--)
    {
        int l0 = 1 << l;
        int l1 = 1 << (l+1);
        for (int j = 0; j < l0; j++)
        {
            if (T[l1 + j*2].value > T[l1 + j*2 + 1].value)
                T[l0 + j] = T[l1 + j*2];
            else
                T[l0 + j] = T[l1 + j*2+1];
        }
    }

    int winner = T[1].list;
    b[winner] += w_k;

    T[SIZE + winner].list = winner;
    if (b[winner] >= alen[winner])
        T[SIZE + winner].value = -MAX_ELEMENT;
    else
        T[SIZE + winner].value = -A[winner][b[winner]] + 1;

    // Now just propagate the winning list
    for (int i = 0; i < k-1; i++)
    {
        int j = winner;
        for (int l = POT-1; l >= 0; l--)
        {
            int l0 = 1 << l;
            int l1 = 1 << (l+1);
            int j_floor = (j >> 1) * 2;
            if (T[l1 + j_floor].value > T[l1 + j_floor + 1].value)
                T[l0 + (j >> 1)] = T[l1 + j_floor];
            else
                T[l0 + (j >> 1)] = T[l1 + j_floor + 1];
            j = j >> 1;
        }
        winner = T[1].list;
        b[winner] += w_k;

        T[SIZE + winner].list = winner;
        if (b[winner] >= alen[winner])
            T[SIZE + winner].value = -MAX_ELEMENT;
        else
            T[SIZE + winner].value = -A[winner][b[winner]] + 1;
   }
}


template <int POT=5>
__device__ void tournament_tree_kth_smallest(int **A, int *alen, int *b, int m, int w_k, int k)
{
    constexpr int SIZE = 1 << POT;
    struct node_t {
        int value;
        int list;
    } T[SIZE * 2];

    for (int i=0; i < m; i++) {
        if (b[i] + w_k > alen[i]) {
            T[SIZE + i].value = MAX_ELEMENT;
            T[SIZE + i].list = -1;
            continue;
        }
        T[SIZE + i].value = A[i][b[i] + w_k - 1] - 1;
        T[SIZE + i].list = i;
    }

    for (int i=m; i < SIZE; i++) {
        T[SIZE + i].value = MAX_ELEMENT;
        T[SIZE + i].list = -1;
    }

    // First iteration, just propagate up the tree
    for (int l = POT-1; l >= 0; l--)
    {
        int l0 = (1 << l);
        int l1 = (1 << (l+1));
        for (int j = 0; j < (1 << l); j++)
        {
            if (T[l1 + j*2].value < T[l1 + j*2 + 1].value)
                T[l0 + j] = T[l1 + j*2];
            else
                T[l0 + j] = T[l1 + j*2+1];
        }
    }

    int winner = T[1].list;
    b[winner] += w_k;

    T[SIZE + winner].list = winner;
    if (b[winner] + w_k > alen[winner])
        T[SIZE + winner].value = MAX_ELEMENT;
    else
        T[SIZE + winner].value = A[winner][b[winner]+w_k-1] - 1;

    // Now just propagate the winning list
    for (int i = 0; i < k-1; i++)
    {
        int j = winner;
        for (int l = POT-1; l >= 0; l--)
        {
            int l0 = 1 << l;
            int l1 = 1 << (l+1);
            int j_floor = (j >> 1) * 2;
            if (T[l1 + j_floor].value < T[l1 + j_floor + 1].value)
                T[l0 + (j >> 1)] = T[l1 + j_floor];
            else
                T[l0 + (j >> 1)] = T[l1 + j_floor + 1];
            j = j >> 1;
        }
        winner = T[1].list;
        b[winner] += w_k;

        T[SIZE + winner].list = winner;
        if (b[winner] + w_k > alen[winner])
            T[SIZE + winner].value = MAX_ELEMENT;
        else
            T[SIZE + winner].value = A[winner][b[winner]+w_k-1] - 1;
   }
}


template <int POT=5>
__device__ void tournament_tree_kth_smallest_reverse(int **A, int *alen, int *b, int m, int w_k, int k)
{
    constexpr int SIZE = 1 << POT;
    struct node_t {
        int value;
        int list;
    } T[SIZE * 2];

    for (int i=0; i < m; i++) {
        if (b[i] - w_k < 0) {
            T[SIZE + i].value = MAX_ELEMENT;
            T[SIZE + i].list = -1;
            continue;
        }
        T[SIZE + i].value = -A[i][b[i] - w_k] + 1;
        T[SIZE + i].list = i;
    }

    for (int i=m; i < SIZE; i++) {
        T[SIZE + i].value = MAX_ELEMENT;
        T[SIZE + i].list = -1;
    }

    // First iteration, just propagate up the tree
    for (int l = POT-1; l >= 0; l--)
    {
        int l0 = (1 << l);
        int l1 = (1 << (l+1));
        for (int j = 0; j < (1 << l); j++)
        {
            if (T[l1 + j*2].value < T[l1 + j*2 + 1].value)
                T[l0 + j] = T[l1 + j*2];
            else
                T[l0 + j] = T[l1 + j*2+1];
        }
    }

    int winner = T[1].list;
    b[winner] -= w_k;

    T[SIZE + winner].list = winner;
    if (b[winner] - w_k < 0)
        T[SIZE + winner].value = MAX_ELEMENT;
    else
        T[SIZE + winner].value = -A[winner][b[winner]-w_k] + 1;

    // Now just propagate the winning list
    for (int i = 0; i < k-1; i++)
    {
        int j = winner;
        for (int l = POT-1; l >= 0; l--)
        {
            int l0 = 1 << l;
            int l1 = 1 << (l+1);
            int j_floor = (j >> 1) * 2;
            if (T[l1 + j_floor].value < T[l1 + j_floor + 1].value)
                T[l0 + (j >> 1)] = T[l1 + j_floor];
            else
                T[l0 + (j >> 1)] = T[l1 + j_floor + 1];
            j = j >> 1;
        }
        winner = T[1].list;
        b[winner] -= w_k;

        T[SIZE + winner].list = winner;
        if (b[winner] - w_k < 0)
            T[SIZE + winner].value = MAX_ELEMENT;
        else
            T[SIZE + winner].value = -A[winner][b[winner]-w_k] + 1;
   }
}


// +/- 1 is because we are zero-based while the Python reference code is 1-based using scipy
__device__ inline int compute_lmax(int **A, int *b, int blen)
{
    int lmax = -MAX_ELEMENT;
    for (int i=0; i < blen; i++) {
        if (b[i] > 0 && (A[i][b[i]-1]-1) > lmax)
            lmax = A[i][b[i]-1] - 1;
    }
    return lmax;
}

__device__ inline int compute_lmax_reverse(int **A, int *alen, int *b, int blen)
{
    int lmax = -MAX_ELEMENT;
    for (int i=0; i < blen; i++) {
        if (b[i] < alen[i] && (-A[i][b[i]]+1) > lmax)
            lmax = -A[i][b[i]]+1;
    }
    return lmax;
}


template <int MAXSIZE=32>
__device__ void row_splitter(int **A, int *alen, int *b, int m, int p)
{
    // assert m < MAXSIZE
    int n_max = -1;
    for (int i=0; i < m; i++) {
        b[i] = 0;
        n_max = (alen[i] > n_max) ? alen[i] : n_max;
    }

    if (p == 0)
        return;

    // Handle short splits
    if (p < m) {
        tournament_tree_kth_smallest(A, alen, b, m, 1, p);
        return;
    }

    int r = ceilf(logf((float)p / m) / logf(2.0));
    int two_r = 1 << r;
    int alpha = n_max / two_r;              // implicit floor
    int n = two_r * (alpha + 1) - 1;
    int k = ceilf((float)p / n * alpha);

    // Initial partition for the recursion
    tournament_tree_kth_smallest(A, alen, b, m, two_r, k);
    int lmax = compute_lmax(A, b, m);

    // r iterative steps
    for (int k=0; k < r; k++)
    {
        int Lsize = 0;
        int w_k = 1 << (r - k - 1);
        int target_size = ceilf((float)p * (n / w_k) / n);

        for (int i=0; i < m; i++)
            Lsize += b[i] / w_k;

        for (int i=0; i < m; i++)
        {
            if (b[i] + w_k > alen[i])
                continue;
            int undecided = A[i][b[i] + w_k - 1] - 1;
            if (undecided < lmax) {
                b[i] += w_k;
                Lsize++;
            }
        }
        if (Lsize > target_size) {
            tournament_tree_kth_largest(A, b, m, w_k, Lsize - target_size);
        }
        if (Lsize < target_size) {
            tournament_tree_kth_smallest(A, alen, b, m, w_k, target_size - Lsize);
        }

        lmax = compute_lmax(A, b, m);
    }
}


template <int MAXSIZE=32>
__device__ void row_splitter_reverse(int **A, int *alen, int *b, int m, int p)
{
    // assert m < MAXSIZE
    int n_max = -1;
    for (int i=0; i < m; i++) {
        b[i] = alen[i];
        n_max = (alen[i] > n_max) ? alen[i] : n_max;
    }

    if (p == 0)
        return;

    // Handle short splits
    if (p < m) {
        tournament_tree_kth_smallest_reverse(A, alen, b, m, 1, p);
        return;
    }

    int r = ceilf(logf((float)p / m) / logf(2.0));
    int two_r = 1 << r;
    int alpha = n_max / two_r;              // implicit floor
    int n = two_r * (alpha + 1) - 1;
    int k = ceilf((float)p / n * alpha);
    
//    printf("m = %d, r = %d, two_r = %d, n_max = %d, alpha = %d, n = %d\n", m, r, two_r, n_max, alpha, n);
//    printf("k = %d\n", k);

    // Initial partition for the recursion
    tournament_tree_kth_smallest_reverse(A, alen, b, m, two_r, k);
//    for (int i=0; i < m; i++)
//        printf("%d ", b[i]);
    /*
    if (threadIdx.x == 0)
    {
    printf("b: [");
    for (int i=0; i < m; i++)
        printf("%d ", b[i]);
    printf("]\n");
    }
    */
    int lmax = compute_lmax_reverse(A, alen, b, m);
//    printf("r = %d, alpha = %d, n = %d, k = %d, n_max = %d, lmax = %d\n", r, alpha, n, k, n_max, lmax);
//    if (threadIdx.x == 0) printf("first lmax = %d\n", lmax);

    // r iterative steps
    for (int k=0; k < r; k++)
    {
        int Lsize = 0;
        int w_k = 1 << (r - k - 1);
        int target_size = ceilf((float)p * (n / w_k) / n);

        for (int i=0; i < m; i++)
            Lsize += (alen[i] - b[i]) / w_k;
//        if (threadIdx.x == 0) printf("Lsize (after decided) = %d\n", Lsize);

        for (int i=0; i < m; i++)
        {
            if (b[i] - w_k < 0)
                continue;
            int undecided = -A[i][b[i] - w_k] + 1;
            if (undecided < lmax) {
                b[i] -= w_k;
                Lsize++;
            }
        }
        /*
        printf("Lsize = %d\n", Lsize);
        printf("pre-boundary: ");
        for (int i=0; i < m; i++)
            printf("%d ", A[i][b[i]-1]-1);
        printf("\n");

        printf("Lsize = %d, target_size = %d\n", Lsize, target_size);
        */
//        if (threadIdx.x == 0) printf("Lsize = %d, target_size = %d\n", Lsize, target_size);
        if (Lsize > target_size) {
//            if (threadIdx.x == 0) printf("moving %d largest from L to H\n", Lsize - target_size);
            tournament_tree_kth_largest_reverse(A, alen, b, m, w_k, Lsize - target_size);
        }
        if (Lsize < target_size) {
//            if (threadIdx.x == 0) printf("moving %d smallest from H to L\n", target_size - Lsize);
            tournament_tree_kth_smallest_reverse(A, alen, b, m, w_k, target_size - Lsize);
        }

        lmax = compute_lmax_reverse(A, alen, b, m);
//        if (threadIdx.x == 0) printf("new lmax = %d\n", lmax);

        /*
        printf("post-boundary: ");
        for (int i=0; i < m; i++)
            printf("%d ", A[i][b[i]-1]-1);
        printf("\n");
        */
    }
}


struct split_t {
    int row;
    int p;
    int out;
    bool reverse;
};


// TODO: compute carry flag
template <int MAXSIZE=32>
__global__ void split_matrix(int *row_ptrs, int *col_idx, int *base, split_t *splits, int nsplits)
{
    int threadId = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (threadId > nsplits)
        return;

    int row = splits[threadId].row;
    int p = splits[threadId].p;
    int *out = base + splits[threadId].out;

    int *A[MAXSIZE];
    int b[MAXSIZE];
    int alen[MAXSIZE];
    int m = row_ptrs[row+1] - row_ptrs[row];

    // assert m < MAXSIZE or handle exception
    for (int i=0; i < m; i++) {
        int brow = col_idx[row_ptrs[row] + i] - 1;
        A[i] = col_idx + row_ptrs[brow];
        alen[i] = row_ptrs[brow+1] - row_ptrs[brow];
    }

    if (splits[threadId].reverse)
        // p is precomputed for reversed rows row_size - split_pt
        row_splitter_reverse(A, alen, b, m, p);
    else
        row_splitter(A, alen, b, m, p);

    for (int i=0; i < m; i++)
        out[i] = b[i];
}


int main(int argc, char **argv)
{
    mgpu::standard_context_t context;

    if (argc < 3) {
        std::cerr << "Usage: " << argv[0] << " <mtx file> <splits>" << std::endl;
        exit(1);
    }

    MatrixMarket matA { argv[1] };
    DeviceMatrix dmA(matA, context);

    std::ifstream ifs(argv[2]);
    std::vector<split_t> splits;
    while (ifs.good()) {
        int row, p, out;
        bool reverse;
        ifs >> row >> p >> out >> reverse;
        if (!ifs.good())
            break;
        splits.push_back({row, p, out, reverse});
    }

    int out_size = splits[splits.size()-1].out;
    std::cout << splits.size() << " splits loaded, size = " << out_size << std::endl;

    mgpu::mem_t<split_t> d_splits = mgpu::to_mem(splits, context);
    // XXX: Size this from the outs above
    mgpu::mem_t<int> d_output(out_size, context);

    float time;
    hipEvent_t start, stop;
    CheckCuda(hipEventCreate(&start));
    CheckCuda(hipEventCreate(&stop));

    hipDeviceSynchronize();

//    split_matrix<<<40, 128>>>(dmA.raw.d_row_ptrs, dmA.raw.d_col_idx, d_output.data(), d_splits.data());
    /*
    cudaThreadSetLimit(hipLimitStackSize, 8192);
    size_t stack_size = 0;
    cudaThreadGetLimit(&stack_size, hipLimitStackSize);
    std::cout << "Stack size: " << stack_size << std::endl;
    CheckCuda(hipGetLastError());
    */
    hipEventRecord(start, 0);
    int n_blocks = (splits.size() / NUM_THREADS) + 1;
    std::cout << "num blocks = " << n_blocks << std::endl;
    for (int i=0; i < NUM_ITERS; i++)
        split_matrix<<<n_blocks, NUM_THREADS>>>(dmA.raw.d_row_ptrs, dmA.raw.d_col_idx, d_output.data(), d_splits.data(), splits.size());
//        split_matrix<<<1, 1>>>(dmA.raw.d_row_ptrs, dmA.raw.d_col_idx, d_output.data(), d_splits.data());
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    std::cout << "Elapsed: " << (time / NUM_ITERS) << " ms" << std::endl;
    hipDeviceSynchronize();
    CheckCuda(hipGetLastError());

    std::vector<int> h_output = mgpu::from_mem(d_output);
    std::ofstream ofs("splits.bin");
    ofs.write((const char *)h_output.data(), out_size * sizeof(int));

    // Load some sample data, and test the tournament tree implementation
}
